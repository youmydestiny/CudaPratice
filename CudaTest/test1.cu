#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h" 
#include ""
#include "hipblas.h"  

#include <time.h>  
#include <iostream>  

// ���������ӷ�kernel��grid��block��Ϊһά
__global__ void add(float* x, float* y, float* z, int n)
{
    // ��ȡȫ������
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    // ����
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
    {
        z[i] = x[i] + y[i];
    }
}

int main()
{
    int N = 1 << 20;
    int nBytes = N * sizeof(float);

    // �����й��ڴ�
    float* x, * y, * z;
    hipMallocManaged((void**)&x, nBytes);
    hipMallocManaged((void**)&y, nBytes);
    hipMallocManaged((void**)&z, nBytes);

    // ��ʼ������
    for (int i = 0; i < N; ++i)
    {
        x[i] = 10.0;
        y[i] = 20.0;
    }

    // ����kernel��ִ������
    dim3 blockSize(256);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x);
    // ִ��kernel
    add << < gridSize, blockSize >>> (x, y, z, N);

    // ͬ��device ��֤�������ȷ����
    hipDeviceSynchronize();
    // ���ִ�н��
    float maxError = 0.0;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(z[i] - 31.0));
    std::cout << "������: " << maxError << std::endl;

    // �ͷ��ڴ�
    hipFree(x);
    hipFree(y);
    hipFree(z);

    return 0;
}