#include "hip/hip_runtime.h" 

#include "hipblas.h"  
 
#include <time.h>  
#include <iostream>  
// ���������ӷ�kernel��grid��block��Ϊһά
__global__ void add(float* x, float* y, float* z, int n)
{
    // ��ȡȫ������
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    // ����
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
    {
        z[i] = x[i] + y[i];
    }
}

int main(){

    int N = 1 << 20;
    int nBytes = N * sizeof(float);
    // ����host�ڴ�
    float* x, * y, * z;
    x = (float*)malloc(nBytes);
    y = (float*)malloc(nBytes);
    z = (float*)malloc(nBytes);

    // ��ʼ������
    for (int i = 0; i < N; ++i)
    {
        x[i] = 10.0;
        y[i] = 20.0;
    }

    // ����device�ڴ�
    float* d_x, * d_y, * d_z;
    hipMalloc((void**)&d_x, nBytes);
    hipMalloc((void**)&d_y, nBytes);
    hipMalloc((void**)&d_z, nBytes);

    // ��host���ݿ�����device
    hipMemcpy((void*)d_x, (void*)x, nBytes, hipMemcpyHostToDevice);
    hipMemcpy((void*)d_y, (void*)y, nBytes, hipMemcpyHostToDevice);
    // ����kernel��ִ������
    dim3 blockSize(256);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x);
    // ִ��kernel
    add <<<gridSize, blockSize>>> (d_x, d_y, d_z, N);

    // ��device�õ��Ľ��������host
    hipMemcpy((void*)z, (void*)d_z, nBytes, hipMemcpyDeviceToHost);

    // ���ִ�н��
    float maxError = 0.0;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(z[i] - 31.0));
    std::cout << "������: " << maxError << std::endl;

    // �ͷ�device�ڴ�
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);
    // �ͷ�host�ڴ�
    free(x);
    free(y);
    free(z);
	return 0;
}